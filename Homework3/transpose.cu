#include "hip/hip_runtime.h"
#include <stdlib.h>
#include <stdio.h>

#include "cuda_utils.h"
#include "timer.c"

typedef float dtype;


__global__ 
void matTrans(dtype* AT, dtype* A, int N)  {
	/* Fill your code here */
    __shared__ float tile[32][32+1];
    
    int x = blockIdx.x * 32 + threadIdx.x;
    int y = blockIdx.y * 32 + threadIdx.y;
    int width = gridDim.x * 32;
  
    for (int j = 0; j < 32; j += 8)
       tile[threadIdx.y+j][threadIdx.x] = A[(y+j)*width + x];
  
    __syncthreads();
  
    x = blockIdx.y * 32 + threadIdx.x;  // transpose block offset
    y = blockIdx.x * 32 + threadIdx.y;
  
    for (int j = 0; j < 32; j += 8) {
       AT[(y+j)*width + x] = tile[threadIdx.x][threadIdx.y + j];
    }
}

void
parseArg (int argc, char** argv, int* N)
{
	if(argc == 2) {
		*N = atoi (argv[1]);
		assert (*N > 0);
	} else {
		fprintf (stderr, "usage: %s <N>\n", argv[0]);
		exit (EXIT_FAILURE);
	}
}


void
initArr (dtype* in, int N)
{
	int i;

	for(i = 0; i < N; i++) {
		in[i] = (dtype) rand () / RAND_MAX;
	}
}

void
cpuTranspose (dtype* A, dtype* AT, int N)
{
	int i, j;

	for(i = 0; i < N; i++) {
		for(j = 0; j < N; j++) {
			AT[j * N + i] = A[i * N + j];
		}
	}
}

int
cmpArr (dtype* a, dtype* b, int N)
{
	int cnt, i;

	cnt = 0;
	for(i = 0; i < N; i++) {
		if(abs(a[i] - b[i]) > 1e-6) cnt++;
	}

	return cnt;
}



void
gpuTranspose (dtype* A, dtype* AT, int N)
{
    struct stopwatch_t* timer = NULL;
    long double t_gpu;

    int pad = 0;
    if (N%32 != 0) {
        pad = 32 - N % 32;
    }
    dim3 dimGrid((N + pad)/32, (N + pad)/32, 1);
    dim3 dimBlock(32, 8, 1);
    // fprintf (stderr, "Finish dim3s\n");

    // Create temp in for padding
    dtype *tempIn = (dtype*) malloc ((N + pad) * (N + pad) * sizeof (dtype));
    for (int i = 0; i < N; ++i) {
        for (int j = 0; j < N; ++j) {
            tempIn[i*(N + pad) + j] = A[i * N + j];
        }
    }
    
    /* Cuda malloc*/
    dtype *idata, *tdata;
    hipMalloc(&idata, (N + pad) * (N + pad) * sizeof (dtype));
    hipMemcpy(idata, tempIn, (N + pad) * (N + pad) * sizeof (dtype), hipMemcpyHostToDevice);
    // fprintf (stderr,  "Finish GPU Mallocs\n");
    hipMalloc(&tdata, (N + pad) * (N + pad) * sizeof (dtype));
    // fprintf (stderr,  "Finish Memcopy1\n");
    /* Setup timers */
    stopwatch_init ();
    timer = stopwatch_create ();

    stopwatch_start (timer);
    /* run your kernel here */
    matTrans<<<dimGrid, dimBlock>>>(tdata, idata, N + pad);
  
    hipDeviceSynchronize ();
    t_gpu = stopwatch_stop (timer);
    // fprintf (stderr,  "Finish matrix Trans\n");
    
    // Undo padding
    dtype* tempOut = (dtype*) malloc ((N + pad) * (N + pad) * sizeof (dtype));
    hipMemcpy(tempOut, tdata, (N + pad) * (N + pad) * sizeof (dtype), hipMemcpyDeviceToHost);
    for (int i = 0; i < N * N; ++i) {
        for (int j = 0; j < N * N; ++j) {
            AT[i * N + j] = tempOut[i * (N + pad) + j];
        }
    }
    free (tempOut);
    free (tempIn);


    fprintf (stderr, "Size N: %d \n", N);
    fprintf (stderr, "GPU transpose: %Lg secs ==> %Lg billion elements/second\n",
           t_gpu, (N * N) / t_gpu * 1e-9 );

    hipFree(idata);
    hipFree(tdata);
    // fprintf (stderr, "Free\n");
}

int 
main(int argc, char** argv)
{
  /* variables */
	dtype *A, *ATgpu, *ATcpu;
  int err;

	int N;

  struct stopwatch_t* timer = NULL;
  long double t_cpu;


	N = -1;
	parseArg (argc, argv, &N);

  /* input and output matrices on host */
  /* output */
  ATcpu = (dtype*) malloc (N * N * sizeof (dtype));
  ATgpu = (dtype*) malloc (N * N * sizeof (dtype));

  /* input */
  A = (dtype*) malloc (N * N * sizeof (dtype));

	initArr (A, N * N);

	/* GPU transpose kernel */
	gpuTranspose (A, ATgpu, N);

  /* Setup timers */
  stopwatch_init ();
  timer = stopwatch_create ();

	stopwatch_start (timer);
  /* compute reference array */
	cpuTranspose (A, ATcpu, N);
  t_cpu = stopwatch_stop (timer);
  fprintf (stderr, "Time to execute CPU transpose kernel: %Lg secs\n",
           t_cpu);

  /* check correctness */
	err = cmpArr (ATgpu, ATcpu, N * N);
	if(err) {
		fprintf (stderr, "Transpose failed: %d\n", err);
	} else {
		fprintf (stderr, "Transpose successful\n");
	}

	free (A);
	free (ATgpu);
	free (ATcpu);

  return 0;
}
